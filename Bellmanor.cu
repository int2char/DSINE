#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include"pathalg.h"
static const int WORK_SIZE =258;
void Bellmanor::copydata(int s,vector<edge>&edges,int nodenum){
};
void Bellmanor::dellocate(){
};
void Bellmanor::allocate(int maxn,int maxedge){
}
void Bellmanor::topsort()
{
};
void Bellmanor::updatE(vector<vector<int>>&tesigns)
{
	esigns=tesigns;
	for(int k=0;k<LY;k++)
		{
			int off=k*nodenum*mm;
			for(int i=0;i<nodenum;i++)
			{
				for(int j=0;j<mm;j++)
					if(j<rus[i].size())
						rudw[off+i*mm+j]=esigns[k][ruw[i][j]];
					else
						rudw[off+i*mm+j]=-1;
			}
		}
	hipMemcpy(dev_rudw,rudw,mm*LY*nodenum*sizeof(int),hipMemcpyHostToDevice);
}
void Bellmanor::updatS(vector<vector<Sot>>&stpair)
{
	L[0]=0;
	L[1]=LY1;
	L[2]=LY2;
	S[0]=stpair[0].size();
	S[1]=stpair[1].size();
	stps=stpair;
	int count=0;
	ncount=L[1]*S[0]+L[2]*S[1];
	memset(d,1,ncount*nodenum*sizeof(int));
	memset(p,-1,ncount*nodenum*sizeof(int));
	for(int k=0;k<L[1];k++)
		{
		for(int j=0;j<stpair[0].size();j++)
			{
			 d[count*nodenum+stpair[0][j].s*NUT]=0;
			 count++;
			}
		}
	for(int k=0;k<L[2];k++)
		{
		for(int j=0;j<stpair[1].size();j++)
			{
			 d[count*nodenum+stpair[1][j].s*NUT]=0;
			 count++;
			}
		}
	Size[0]=nodenum*L[1]*S[0];
	Size[1]=nodenum*L[2]*S[1];
	hipMemcpy(dev_d,d,ncount*nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_p,p,ncount*nodenum*sizeof(int),hipMemcpyHostToDevice);
}
void Bellmanor::init(pair<vector<edge>,vector<vector<int>>>ext,vector<pair<int,int>>stpair,int _nodenum)
{
	//cout<<"init bellmanor"<<endl;
	nodenum=_nodenum;
	edges=ext.first;
	esigns=ext.second;
	stp=stpair;
	W=WD+1;
	//st=new int[edges.size()*LY];
	//te=new int[edges.size()*LY];
	d=new int[nodenum*LY*YE];
	has=new int[nodenum*LY*YE];
	p=new int[nodenum*LY*YE];
	w=new int[edges.size()*LY];
	m1=new int;
	m2=new int;
	*m1=0,*m2=0;
	esignes=new int[edges.size()*LY];
	vector<vector<int>>nein(nodenum*LY,vector<int>());
	neibn=nein;
	vector<vector<int>>neie(nodenum,vector<int>());
	vector<vector<int>>rs(nodenum,vector<int>());
	vector<vector<int>>rw(nodenum,vector<int>());
	rus=rs;
	ruw=rw;
	for(int i=0;i<edges.size();i++)
		{
			int s=edges[i].s;
			int t=edges[i].t;
			rus[t].push_back(s);
			ruw[t].push_back(i);
			neibn[s].push_back(t);
			neie[s].push_back(i);
		}
	mm=0;
	for(int i=0;i<rus.size();i++)
		if(rus[i].size()>mm)mm=rus[i].size();
	rudu=new int[nodenum*mm*LY];
	rudw=new int[nodenum*mm*LY];
	rid=new int[nodenum*mm*LY];
	for(int k=0;k<LY;k++)
		{
		int off=k*nodenum*mm;
		for(int i=0;i<nodenum;i++)
		{
			for(int j=0;j<mm;j++)
				if(j<rus[i].size())
					rudu[off+i*mm+j]=rus[i][j];
				else
					rudu[off+i*mm+j]=INT_MAX;
			for(int j=0;j<mm;j++)
				if(j<rus[i].size())
					{	
						rudw[off+i*mm+j]=esigns[k][ruw[i][j]];
						rid[off+i*mm+j]=ruw[i][j];
					}
				else
					{
						rudw[off+i*mm+j]=-1;
						rid[off+i*mm+j]=-1;
					}
		}
		}
	int count=0;
	/*for(int k=0;k<LY;k++)
		for(int i=0;i<nodenum;i++)
			for(int j=0;j<neibn[i].size();j++)
			{
				st[count]=i;
				if(esigns[k][neie[i][j]]<0)
					te[count]=i;
				else
					te[count]=neibn[i][j];
				count++;
			}*/
	int cc=0;
	for(int k=0;k<LY;k++)
		for(int i=0;i<edges.size();i++)
			w[cc++]=esigns[k][i];
	//hipMalloc((void**)&dev_st,LY*edges.size()*sizeof(int));
	//hipMalloc((void**)&dev_te,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_d,YE*LY*nodenum*sizeof(int));
	hipMalloc((void**)&dev_p,YE*LY*nodenum*sizeof(int));
	hipMalloc((void**)&dev_w,LY*edges.size()*sizeof(int));
	//hipMalloc((void**)&dev_m1,sizeof(int));
	//hipMalloc((void**)&dev_m2,sizeof(int));
	hipMalloc((void**)&dev_rudu,mm*LY*nodenum*sizeof(int));
	hipMalloc((void**)&dev_rudw,mm*LY*nodenum*sizeof(int));
	hipMalloc((void**)&dev_rid,mm*LY*nodenum*sizeof(int));
	//hipMalloc((void**)&dev_ruid,mm*LY*nodenum*sizeof(int));
	//hipMemcpy(dev_te,te,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	//hipMemcpy(dev_st,st,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_w,w,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_rudu,rudu,mm*LY*nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_rudw,rudw,mm*LY*nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_rid,rid,mm*LY*nodenum*sizeof(int),hipMemcpyHostToDevice);
	//hipMemcpy(dev_ruid,ruid,mm*LY*nodenum*sizeof(int),hipMemcpyHostToDevice);
	//hipMemcpy(dev_m1,m1,sizeof(int),hipMemcpyHostToDevice);
	//hipMemcpy(dev_m2,m2,sizeof(int),hipMemcpyHostToDevice);
};
Bellmanor::Bellmanor():L(PC+1,0),S(PC,0),NF(PC,0),Size(2,0)
{
};
__global__ void bellmandu(int *rudu,int*rudw,int *d,int*p,int N,int size,int sizeoff,int leveloff,int ye,int ly,int mm)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=size)return;
	int lyy=i/(ye*N);
	int yee=(i%(ye*N))/N;
	int off=lyy*N*ye+yee*N+sizeoff;
	int roff=(i%N+(lyy+leveloff)*N)*mm;
	i+=sizeoff;
	int dm=d[i];
	for(int k=0;k<mm;k++)
		if(rudu[roff+k]<INT_MAX)
		{
			int node=rudu[roff+k]+off;
			if(rudw[roff+k]<0)continue;
			if(dm>d[node]+rudw[roff+k])
				dm=d[node]+rudw[roff+k];
		}
	if(d[i]>dm)
		d[i]=dm;
}
__global__ void bellmancolor(int *rudu,int*rudw,int*rid,int *d,int*p,int N,int size,int sizeoff,int leveloff,int ye,int ly,int mm)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=size)return;
	int lyy=i/(ye*N);
	int yee=(i%(ye*N))/N;
	int off=lyy*N*ye+yee*N+sizeoff;
	int roff=(i%N+(lyy+leveloff)*N)*mm;
	i+=sizeoff;
	int dm=d[i];
	int mark=-1;
	for(int k=0;k<mm;k++)
		if(rudu[roff+k]<INT_MAX)
		{
			int node=rudu[roff+k]+off;
			if(rudw[roff+k]<0)continue;
			if(dm==d[node]+rudw[roff+k])
				{mark=rid[roff+k];break;}
		}
	p[i]=mark;
}
vector<vector<Rout>> Bellmanor::routalg(int s,int t,int bw)
{
	//cout<<"inbellman"<<endl;
	int kk=1;
	time_t start,end;
	start=clock();
	hipStream_t stream0;
	hipStreamCreate(&stream0);
	hipStream_t stream1;
	hipStreamCreate(&stream1);
	for(int i=0;i<WD+1;i++)
	{
		bellmandu<<<Size[0]/512+1,512,0>>>(dev_rudu,dev_rudw,dev_d,dev_p,nodenum,Size[0],0,0,S[0],L[1],mm);
		bellmandu<<<Size[1]/512+1,512,0>>>(dev_rudu,dev_rudw,dev_d,dev_p,nodenum,Size[1],Size[0],L[1],S[1],L[2],mm);
	}
	bellmancolor<<<Size[0]/512+1,512,0>>>(dev_rudu,dev_rudw,dev_rid,dev_d,dev_p,nodenum,Size[0],0,0,S[0],L[1],mm);
	bellmancolor<<<Size[1]/512+1,512,0>>>(dev_rudu,dev_rudw,dev_rid,dev_d,dev_p,nodenum,Size[1],Size[0],L[1],S[1],L[2],mm);
	hipStreamSynchronize(stream1);
	hipStreamSynchronize(stream0);
	hipMemcpy(d,dev_d,ncount*nodenum*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(p,dev_p,ncount*nodenum*sizeof(int),hipMemcpyDeviceToHost);
	end=clock();
	vector<vector<Rout>>result(2,vector<Rout>());
	vector<int>LL(3,0);
	LL=L;
	LL[2]+=LL[1];
	int count=0;
	for(int y=1;y<PC+1;y++)
		for(int k=LL[y-1];k<LL[y];k++)
		{	
			for(int l=0;l<stps[y-1].size();l++)
			{	
				int offf=count*nodenum;
				int s=stps[y-1][l].s*NUT;
				vector<int>ters=stps[y-1][l].ters;
				for(int i=0;i<ters.size();i++)
				{
					int id=stps[y-1][l].mmpid[ters[i]];
					int hop=0;
					int tt=ters[i];
					int min=INF;
					int prn=-1;
					for(int i=1;i<W;i++)
						{
						if(d[offf+tt*W+i]<min)
							{	
								min=d[offf+tt*W+i];
								prn=offf+tt*W+i;
							}
						}
					int node=prn-offf;
					if(prn<0)continue;
					Rout S(s,node,id,min,offf,k);
					result[y-1].push_back(S);
				}
				count++;
			}
		}
	//cout<<"GPU time is : "<<end-start<<endl;
	return result;
};

/*
__global__ void bellmanhigh(int *st,int *te,int *d,int *has,int *w,int E,int N,int size,int *m,int round,int Leveloff,int numoff,int ye,int ly)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>size)return;	
	int eid=(i%(E*ly));
	int eeid=eid+Leveloff;
	int s=st[eeid],t=te[eeid],weight=w[eeid];
	if(weight<0)return;
	int off=(i/(E*ly))*N+(eid/E)*N*ye+numoff;
	//if(has[s+off]<round-1)return;
	if(d[s+off]+weight<d[t+off])  
		{
			d[t+off]=weight+d[s+off];
			//has[t+off]=round;
			*m=1;
		}
}*/
/*__global__ void color(int *st,int *te,int *d,int *pre,int *has,int *w,int E,int N,int size,int round,int Leveloff,int numoff,int ye,int ly)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>size)return;	
	int eid=(i%(E*ly));
	int eeid=eid+Leveloff;
	int s=st[eeid],t=te[eeid],weight=w[eeid];
	if(weight<0)return;
	int off=(i/(E*ly))*N+(eid/E)*N*ye+numoff;
	//if(has[s+off]<round-1)return;
	if(d[s+off]+weight==d[t+off])
		pre[t+off]=s+off;
}*/
/*m1=1;
	*m2=1;
	int round=1;
	cout<<"fuck wx!"<<endl;
	int flag1=0,flag2=0;
	int cc=0;
	while(*m2==1||*m1==1)
	{
		*m2=0,*m1=0;
		hipMemcpyAsync(dev_m2,m2,sizeof(int),hipMemcpyHostToDevice,stream1);
		bellmanhigh<<<size[1]/1024+1,1024,0,stream1>>>(dev_st,dev_te,dev_d,dev_has,dev_w,edges.size(),nodenum,size[1],dev_m2,round,leveloff[1],nodeoff[1],S[1],L[1]);
		hipMemcpyAsync(dev_m1,m1,sizeof(int),hipMemcpyHostToDevice,stream0);
		bellmanhigh<<<size[0]/1024+1,1024,0,stream0>>>(dev_st,dev_te,dev_d,dev_has,dev_w,edges.size(),nodenum,size[0],dev_m2,round,leveloff[0],nodeoff[0],S[0],L[0]);
		color<<<size[1]/1024+1,1024,0,stream1>>>(dev_st,dev_te,dev_d,dev_p,dev_has,dev_w,edges.size(),nodenum,size[1],round,leveloff[1],nodeoff[1],S[1],L[1]);
		hipMemcpyAsync(m2,dev_m2,sizeof(int),hipMemcpyDeviceToHost,stream1);
		color<<<size[0]/1024+1,1024,0,stream0>>>(dev_st,dev_te,dev_d,dev_p,dev_has,dev_w,edges.size(),nodenum,size[0],round,leveloff[0],nodeoff[0],S[0],L[0]);
		hipMemcpyAsync(m1,dev_m1,sizeof(int),hipMemcpyDeviceToHost,stream0);
		hipStreamSynchronize(stream1);
		hipStreamSynchronize(stream0);
	}*/