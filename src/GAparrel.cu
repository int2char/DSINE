#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include"Graph.h"
#include "service.h"
#include"taskPath.h"
#include"valuemark.h"
#include"hiprand/hiprand_kernel.h"
#include"iostream"
#include <fstream>
#include"const.h"
#include<math.h>
#include"BFS.h"
#include"GAparrel.h"
#include<time.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include"PathArrange.h"
__global__ void PathChoose(int T,int M,int W,double *x,double *y,double*u,double*f,int*paths)
{
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	int tid=threadIdx.x;
	if(id>=T*W)return;
	int d=id/W;
	int k=id%W;
	__shared__ double price[256];
	price[tid]=0;
	int off=d*M*W+k*M;
	for(int i=0;i<M;i++)
		{
		if(paths[off+i]<0)break;
		price[tid]+=u[paths[off+i]];
		}
	if(price[tid]<=0)price[tid]=DBL_MAX;
	if(k==0)
	{
		double gu=price[tid];
		int bid=0;
		for(int i=1;i<W;i++)
			if(price[tid+i]<gu)
				{
				gu=price[tid+i];
				bid=i;
				}
		y[d]+=(0.05/12)*(x[d]-y[d]);
		x[d]=pow(pow(y[d],-6)/price[tid+bid],10)*y[d];
		int off=d*M*W+bid*M;
		for(int i=0;i<M;i++)
			{
			if(paths[off+i]<0)break;
			f[paths[off+i]]+=x[d];
			}
	}
}
__global__ void changeU(int E,double*u,double*f)
{
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	if(id>=E)return;
	u[id]+=u[id]*0.05*(f[id]-100.0)/100.0;
	f[id]=0;
}
__global__ void Sum(int T,double*x,double*sum)
{
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	if(id>T/2)return;
	int bid=blockIdx.x;
	int off=bid*1024;
	int left=min(T-512*bid,512);
	int tid=threadIdx.x;
	__shared__ double tsum[256];
	tsum[tid]=pow(x[tid+off],-5)/5;
	if(id>=T/2)return;
	tsum[tid]+=pow(x[tid+off+(left+1)/2],-5)/5;
	for(int s=(left+1)/2;s>1;s=(s+1)/2)
	{
		if(tid<s/2)
			tsum[tid]+=tsum[tid+(s+1)/2];
		__syncthreads();
	}
	__syncthreads();
	if(tid==0)
		sum[bid]=tsum[0];
		
}
void NewGAParrel::Cudamalloc(){
	cout<<"m is"<<M<<endl;
	hipMalloc((void**)&dev_x, (T+1)*sizeof(double));
	hipMalloc((void**)&dev_y, T*sizeof(double));
	hipMalloc((void**)&dev_u, E*sizeof(double));
	hipMalloc((void**)&dev_f, E*sizeof(double));
	hipMalloc((void**)&dev_paths, T*M*W*sizeof(int));
	hipMalloc((void**)&dev_sum, (T/512+1)*sizeof(double));
	hipMemcpy(dev_x,x, sizeof(double)*(T+1),hipMemcpyHostToDevice);
	hipMemcpy(dev_y,y, sizeof(double)*T,hipMemcpyHostToDevice);
	hipMemcpy(dev_u,u, sizeof(double)*E,hipMemcpyHostToDevice);
	hipMemcpy(dev_f,f, sizeof(double)*E,hipMemcpyHostToDevice);
	hipMemcpy(dev_paths,paths,sizeof(int)*T*W*M,hipMemcpyHostToDevice);
	hipMemcpy(dev_sum,sum,sizeof(double)*(T/512+1),hipMemcpyHostToDevice);
}
vector<pair<string,float> > NewGAParrel::GAsearch(){
	Cudamalloc();
	cout<<"m is "<<M<<endl;
	for(int i=0;i<100000;i++)
	{
		PathChoose<< <T*W/256+1,256>> >(T,M,W,dev_x,dev_y,dev_u,dev_f,dev_paths);
		hipMemcpy(f,dev_f, sizeof(double)*E,hipMemcpyDeviceToHost);
		changeU<< <E/512+1,512>> >(E,dev_u,dev_f);
		Sum<<<T/256+1,256>>>(T,dev_x,dev_sum);
		hipMemcpy(sum,dev_sum,sizeof(double)*(T/1024+1),hipMemcpyDeviceToHost);
		cout<<"sum o is: "<<sum[0]<<endl;
	}
	cout<<"what happened"<<endl;
}
