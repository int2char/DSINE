#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include"Graph.h"
#include "service.h"
#include"taskPath.h"
#include"valuemark.h"
#include"hiprand/hiprand_kernel.h"
#include"iostream"
#include <fstream>
#include"const.h"
#include<math.h>
#include"BFS.h"
#include"GAparrel.h"
#include<time.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include"PathArrange.h"
__device__ double Add(double* address, double val)
{
    unsigned long long int* address_as_ull =
                             (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}
__global__ void getpath(Ldge*edge,int*p,double*x,double*f,double*u,int*s,int*t,int NN,int T)
{
	int id=threadIdx.x + blockDim.x*blockIdx.x;
	if(id>=T)return;
	int pre=t[id];
	int off=NN*s[id];
	int ss=s[id];
	double v=0;
	int tm=-1;
	/*while(pre!=ss)
		{
			tm=p[pre];
			v+=u[tm];
			pre=edge[tm].head;
			Add(&f[tm],1.0);
		}*/
	Add(&f[id],1.0);
	x[id]=pow(v,-(1.0/6));
}
__global__ void ChangePameterC(int*p,double*d,int n){
	int tid = blockIdx.y;
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	if (i>=n||tid >=n)return;
	int biao = tid*n + i;
	d[biao] = (i == tid) ? 0.0:DBL_MAX/2;
	p[biao] = -1;
}
__global__ void bellmanHigh(Ldge*edge, int*m, double*c, int*p, double*u,int E,int NN)
{
	int tid = blockIdx.y;
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i >=E)return;
	int head = edge[i].head;
	int tail = edge[i].tail;
	int biao = tid*NN+head;
	double val = c[tid*NN+tail]+u[i];
	if (c[biao] >val){
		*m = 1;
		c[biao] = val;
	}
	//*m=1;
}
__global__ void color(Ldge *edge, int *m, double*c,int*p,double*u,int E,int NN){

	int tid = blockIdx.y;
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i >=E)return;
	int head = edge[i].head;
	int tail = edge[i].tail;
	int biao = tid*NN+head;
	double val = c[tail+tid*NN]+u[i];
	if (c[biao] == val){
		p[biao] = tid;
	}
}

__global__ void changeU(int E,double*u,double*f)
{
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	if(id>=E)return;
	u[id]+=u[id]*0.05*(f[id]-100.0)/100.0;
	f[id]=0;
}
__global__ void Sum(int T,double*x,double*sum)
{
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	if(id>T/2)return;
	int bid=blockIdx.x;
	int off=bid*1024;
	int left=min(T-512*bid,512);
	int tid=threadIdx.x;
	__shared__ double tsum[256];
	tsum[tid]=pow(x[tid+off],-5)/5;
	if(id>=T/2)return;
	tsum[tid]+=pow(x[tid+off+(left+1)/2],-5)/5;
	for(int s=(left+1)/2;s>1;s=(s+1)/2)
	{
		if(tid<s/2)
			tsum[tid]+=tsum[tid+(s+1)/2];
		__syncthreads();
	}
	__syncthreads();
	if(tid==0)
		sum[bid]=tsum[0];
		
}
void NewGAParrel::Cudamalloc(){
	cout<<"m is"<<M<<endl;
	hipMalloc((void**)&dev_x, (T+1)*sizeof(double));
	hipMalloc((void**)&dev_y, T*sizeof(double));
	hipMalloc((void**)&dev_u, E*sizeof(double));
	hipMalloc((void**)&dev_f, E*sizeof(double));
	hipMalloc((void**)&dev_sum, (T/512+1)*sizeof(double));
	hipMalloc((void**)&dev_d, NN*NN*sizeof(double));
	hipMalloc((void**)&dev_p, NN*NN*sizeof(int));
	hipMalloc((void**)&dev_edge, E*sizeof(Ldge));
	hipMalloc((void**)&dev_s, T*sizeof(int));
	hipMalloc((void**)&dev_t, T*sizeof(int));
	hipMalloc((void**)&dev_m, sizeof(int));
	hipMemcpy(dev_x,x, sizeof(double)*(T+1),hipMemcpyHostToDevice);
	hipMemcpy(dev_y,y, sizeof(double)*T,hipMemcpyHostToDevice);
	hipMemcpy(dev_u,u, sizeof(double)*E,hipMemcpyHostToDevice);
	hipMemcpy(dev_f,f, sizeof(double)*E,hipMemcpyHostToDevice);
	hipMemcpy(dev_sum,sum,sizeof(double)*(T/512+1),hipMemcpyHostToDevice);
	hipMemcpy(dev_d,d,sizeof(double)*NN*NN,hipMemcpyHostToDevice);
	hipMemcpy(dev_p,p,sizeof(int)*NN*NN,hipMemcpyHostToDevice);
	hipMemcpy(dev_edge,edge,sizeof(Ldge)*E,hipMemcpyHostToDevice);
	hipMemcpy(dev_s,s,sizeof(int)*T,hipMemcpyHostToDevice);
	hipMemcpy(dev_t,t,sizeof(int)*T,hipMemcpyHostToDevice);
	hipMemcpy(dev_m,mm,sizeof(int),hipMemcpyHostToDevice);
}
void NewGAParrel::GAsearch(){
	Cudamalloc();
	cout<<"ajsgkvksafuaqd"<<endl;
	time_t begin=clock();
	*mm=0;
	for(int i=0;i<100;i++)
	{
		dim3 blocksq(NN/64+1, NN*NN);
		ChangePameterC << <blocksq,64>> >(dev_p, dev_d,NN);
		dim3 blocks_square(E/256+1,NN);
		int cc=0;
		do{
			if(cc%8==0)
				{*mm=0;
				hipMemcpy(dev_m,mm, sizeof(int),hipMemcpyHostToDevice);}
			bellmanHigh << <blocks_square,256>> >(dev_edge, dev_m, dev_d, dev_p, dev_u,E,NN);
			if(cc%8==0)
				hipMemcpy(mm,dev_m, sizeof(int), hipMemcpyDeviceToHost);
			cc++;
		} while (*mm);
		//cout<<cc<<endl;
		color<< <blocks_square,256>> >(dev_edge, dev_m, dev_d, dev_p, dev_u,E,NN);
		//hipMemcpy(d,dev_d,sizeof(double)*NN*NN,hipMemcpyDeviceToHost);
		//for(int i=0;i<NN-1;i++)
			//cout<<d[i*NN+i+1]<<endl;
		//getpath<<<T/256+1,256>>>(dev_edge,dev_p,dev_x,dev_f,dev_u,dev_s,dev_t,NN,T);
		//hipMemcpy(f,dev_f,sizeof(double)*E,hipMemcpyDeviceToHost);
		/*for(int i=0;i<E;i++)
			cout<<f[i]<<" ";
		cout<<endl;*/
		//changeU<< <E/512+1,512>> >(E,dev_u,dev_f);
		//Sum<<<T/256+1,256>>>(T,dev_x,dev_sum);
		//hipMemcpy(sum,dev_sum,sizeof(double)*(T/1024+1),hipMemcpyDeviceToHost);*/
	}
	time_t end=clock();
	cout<<end-begin<<endl;
	cout<<"what happened"<<endl;
}
